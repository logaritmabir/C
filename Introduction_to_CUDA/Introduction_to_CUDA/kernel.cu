﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

const int array_size = 32*32;
const int allocation_size = array_size * sizeof(int);

void* cpu_allocation;
void* gpu_allocation;

int a;

void cpu_set_array() {
	int* cpu_int32 = (int*)cpu_allocation;

	for (int i = 0; i < array_size; i++) {
		cpu_int32[i] = i;
	}
}
void cpu_alloc(){
	cpu_allocation = (void*)malloc(allocation_size); // malloc geri dönüş değeri zaten void * tipinde olacak,sadece paşa keyfim yazmak istedi
}
void cpu_free() {
	free(cpu_allocation);
}
void gpu_alloc() {
	hipError_t result = hipMalloc(&gpu_allocation,allocation_size); // Imagine like system call.
	assert(result == 0);
}
void gpu_free() {
	hipError_t result = hipFree(gpu_allocation);
	assert(result == 0);
}
void cpu_mem_to_gpu_mem() {
	hipError_t result = hipMemcpy(gpu_allocation,cpu_allocation,allocation_size,hipMemcpyHostToDevice);
	assert(result == 0);
}
void gpu_mem_to_cpu_mem() {
	hipError_t result = hipMemcpy(cpu_allocation, gpu_allocation, allocation_size, hipMemcpyDeviceToHost);
	assert(result == 0);
}

__global__ void gpu_add(int* gpu_numbers) {
	int thread_index_in_block = threadIdx.x; // bölüt içerisinde izleğin indisi
	int block_index_in_grid = blockIdx.x;	// ızgara içerisinde bölütün indisi
	//int thread_count_in_one_block = blockDim.x;		// bölütün içerisinde kaç tane izlek var
	//int number_of_blocks_in_grid = gridDim.x;	// ızgara kaç tane bölüt içeriyor

	int id = blockDim.x * blockIdx.x + threadIdx.x;// her bir izleğe eşsiz bir indis oluşturma

	if (thread_index_in_block == 29 && block_index_in_grid == 0  ) {
		printf("");
	}
	gpu_numbers[id] *= 3;

}

void print_number() {

	int* numbers = (int*)cpu_allocation;
	for (int i = 0; i < array_size; i++) {
		printf("%d \t %d\n", i, numbers[i]);
	}
}
int main()
{
	cpu_alloc();
	gpu_alloc(); // en son hangisini tahsis ettiysen ilk onu boşalt

	cpu_set_array();

	cpu_mem_to_gpu_mem();

	int block_dimension = 64; //bu sayı bir bölüt içerisinde çalışacak izlek sayısını temsil etmektedir.bunu kendim ayarlıyorum şu an için.
	int grid_dimension = array_size / block_dimension; //Özdevimli olarak bu diziyi eş zamanlı atayacak ızgara bölüt sayısını ayarlıyorum.



	gpu_add <<<grid_dimension,block_dimension >>> ((int*)(gpu_allocation)); // bir bölüt(block) en fazla 1024 izlek içerebilir,	number of blocks , number of threads per block 
	hipError_t result  = hipDeviceSynchronize();
	assert(result == hipSuccess);

	gpu_mem_to_cpu_mem();

	print_number();


	gpu_free();
	cpu_free();
}

//CUDA'da bellek tahsisi maliyetli bir işlem olduğu için olduğunca algoritma düzeyinde bellek sorunları çözülmelidir.
